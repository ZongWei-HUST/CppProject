#include <hip/hip_runtime.h>


#include <iostream>
using std::cout;
using std::endl;

__global__ void gpuAdd(int* d_a, int* d_b, int* d_c) {
  *d_c = *d_a + *d_b;
  printf("This is block: %d, thread: %d\n", blockIdx.x, threadIdx.x);
}

int main() {
  // 为CPU/GPU分配内存
  int h_c, h_a, h_b;
  int *d_c, *d_a, *d_b;
  h_a = 1;
  h_b = 4;
  hipMalloc((void**)&d_c, sizeof(int));  // 为GPU分配内存
  hipMalloc((void**)&d_a, sizeof(int));
  hipMalloc((void**)&d_b, sizeof(int));
  // 将数据从CPU拷贝到GPU
  hipMemcpy(d_c, &h_c, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_a, &h_a, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_b, &h_b, sizeof(int), hipMemcpyHostToDevice);
  // 内核调用
  gpuAdd<<<8, 2>>>(d_a, d_b, d_c);
  // 将数据从GPU拷贝到CPY
  hipMemcpy(&h_c, d_c, sizeof(int), hipMemcpyDeviceToHost);
  printf("1 + 4 = %d\n", h_c);
  // 释放主机和设备内存
  hipFree(d_c);
  hipFree(d_a);
  hipFree(d_b);
  return 0;
}