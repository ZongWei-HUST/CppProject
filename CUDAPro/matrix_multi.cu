// Matrix multiplication using shared and non shared kernal
#include <hip/hip_runtime.h>

#include <math.h>

#include <iostream>

#include "stdio.h"
#define TILE_SIZE 2

// Matrix multiplication using non shared kernel
__global__ void gpu_Matrix_Mul_nonshared(float *d_a, float *d_b, float *d_c,
                                         const int size) {
  int row, col;
  col = TILE_SIZE * blockIdx.x + threadIdx.x;
  row = TILE_SIZE * blockIdx.y + threadIdx.y;
  //   printf("col: %d, row: %d, \n", col, row);
  printf("blockIdx: %d, threadIdx: %d, \n", blockIdx.y, threadIdx.y);

  for (int k = 0; k < size; k++) {
    d_c[row * size + col] += d_a[row * size + k] * d_b[k * size + col];
  }
}

// Matrix multiplication using shared kernel
__global__ void gpu_Matrix_Mul_shared(float *d_a, float *d_b, float *d_c,
                                      const int size) {
  int row, col;
  // Defining Shared Memory
  __shared__ float shared_a[TILE_SIZE][TILE_SIZE];
  __shared__ float shared_b[TILE_SIZE][TILE_SIZE];
  col = TILE_SIZE * blockIdx.x + threadIdx.x;
  row = TILE_SIZE * blockIdx.y + threadIdx.y;

  for (int i = 0; i < size / TILE_SIZE; i++) {
    shared_a[threadIdx.y][threadIdx.x] =
        d_a[row * size + (i * TILE_SIZE + threadIdx.x)];
    shared_b[threadIdx.y][threadIdx.x] =
        d_b[(i * TILE_SIZE + threadIdx.y) * size + col];
    __syncthreads();
    for (int j = 0; j < TILE_SIZE; j++)
      d_c[row * size + col] +=
          shared_a[threadIdx.y][j] * shared_b[j][threadIdx.x];
    __syncthreads();
  }
}

// main routine
int main() {
  const int size = 4;
  // Define Host Array
  float h_a[size][size], h_b[size][size], h_result[size][size];
  // Defining device Array
  float *d_a, *d_b, *d_result;
  // Initialize host Array
  for (int i = 0; i < size; i++) {
    for (int j = 0; j < size; j++) {
      h_a[i][j] = i;
      h_b[i][j] = j;
    }
  }

  hipMalloc((void **)&d_a, size * size * sizeof(int));
  hipMalloc((void **)&d_b, size * size * sizeof(int));
  hipMalloc((void **)&d_result, size * size * sizeof(int));

  // copy host array to device array

  hipMemcpy(d_a, h_a, size * size * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, size * size * sizeof(int), hipMemcpyHostToDevice);

  // Define grid and block dimensions
  dim3 dimGrid(size / TILE_SIZE, size / TILE_SIZE, 1);
  dim3 dimBlock(TILE_SIZE, TILE_SIZE, 1);
  //   gpu_Matrix_Mul_nonshared<<<dimGrid, dimBlock>>>(d_a, d_b, d_result,
  //   size);

  gpu_Matrix_Mul_shared<<<dimGrid, dimBlock>>>(d_a, d_b, d_result, size);

  hipMemcpy(h_result, d_result, size * size * sizeof(int),
             hipMemcpyDeviceToHost);
  printf("The result of Matrix multiplication is: \n");

  for (int i = 0; i < size; i++) {
    for (int j = 0; j < size; j++) {
      printf("%f   ", h_result[i][j]);
    }
    printf("\n");
  }
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_result);
  return 0;
}