#include <hip/hip_runtime.h>


#include <iostream>

#define N 5
#define si sizeof(int)
#define sf sizeof(float)

__global__ void gpuSquare(float *d_in, float *d_out) {
  int tid = threadIdx.x;
  float tmp = d_in[tid];
  d_out[tid] = tmp * tmp;
}

int main() {
  float h_in[N], h_out[N];
  float *d_in, *d_out;
  for (int i = 0; i < N; i++) {
    h_in[i] = i;
  }
  hipMalloc((void **)&d_in, sf * N);
  hipMalloc((void **)&d_out, sf * N);
  hipMemcpy(d_in, h_in, sf * N, hipMemcpyHostToDevice);
  gpuSquare<<<1, N>>>(d_in, d_out);
  hipMemcpy(h_out, d_out, sf * N, hipMemcpyDeviceToHost);
  for (int i = 0; i < N; i++) {
    printf("The square of %f is %f\n", h_in[i], h_out[i]);
  }
  hipFree(d_in);
  hipFree(d_out);
  return 0;
}